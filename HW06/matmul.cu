#include "hip/hip_runtime.h"
#include "matmul.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel for matrix multiplication
// Computes matrix C as the product of matrices A and B, all stored in row-major format
__global__ void matmul_kernel(const float* A, const float* B, float* C, size_t n) {
    // Calculate the global position of the current thread
    size_t pos = blockIdx.x * blockDim.x + threadIdx.x;
    size_t size = n * n;

    // Ensure the thread is within bounds
    if (pos < size) {
	float value = 0.0f;
	int r = pos / n; // Row index
	int c = pos % n; // Column index

        // Perform the dot product for the current element
        for (size_t k = 0; k < n; k++) {
            value += A[r * n + k] * B[k * n + c];
        }
        C[r * n + c] = value;
    }
}	
// Host function to perform matrix multiplication using CUDA
// Transfers data between host and device, launches kernel, and retrieves the result
void matmul(const float* A, const float* B, float* C, size_t n, unsigned int threads_per_block) {
    // Calculate the required number of blocks to cover all elements
    size_t num_block = (threads_per_block - 1 + n * n) / threads_per_block;

    // Device pointers for matrices
    float *device_A, *device_B, *device_C;
    // Allocate memory on the GPU for matrices
    hipMalloc((void**)&device_A, n * n * sizeof(float));
    hipMalloc((void**)&device_B, n * n * sizeof(float));
    hipMalloc((void**)&device_C, n * n * sizeof(float));
	
    // Copy input matrices from host to device
    hipMemcpy(device_A, A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_B, B, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(device_C, 0, n*n*sizeof(float));

    // Launch the matrix multiplication kernel
    matmul_kernel<<<num_block, threads_per_block>>>(device_A, device_B, device_C, n);

    // Wait for the kernel to finish execution
    hipDeviceSynchronize();

    // Copy the result matrix back to the host
    hipMemcpy(C, device_C, n * n * sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(device_A);
    hipFree(device_B);
    hipFree(device_C);
}
